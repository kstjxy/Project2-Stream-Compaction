#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernUpSweep(int nPow2, int d, int* data) {
            int i = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = 1 << (d + 1);
            int bi = i * stride + (stride - 1);
            if (bi >= nPow2) return;

            int ai = bi - (1 << d);
            data[bi] += data[ai];
        }

        __global__ void kernDownSweep(int nPow2, int d, int* data) {
            int i = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = 1 << (d + 1);
            int bi = i * stride + (stride - 1);
            if (bi >= nPow2) return;

            int ai = bi - (1 << d);
            int t = data[ai];
            data[ai] = data[bi];
            data[bi] += t;
        }

        static void scanInPlace(int* devData, int nPow2) {
            if (nPow2 <= 0) return;

            const int BLOCK_SIZE = 128;
            int levels = ilog2ceil(nPow2);  // = log2(nPow2) since nPow2 is power-of-two

            // Up-sweep
            for (int d = 0; d < levels; ++d) {
                int numOps = nPow2 >> (d + 1);
                dim3 block(BLOCK_SIZE);
                dim3 grid((numOps + BLOCK_SIZE - 1) / BLOCK_SIZE);
                kernUpSweep << <grid, block >> > (nPow2, d, devData);
                checkCUDAError("kernUpSweep");
            }

            // Set root to 0 (exclusive) WITHOUT another kernel
            // (only two helper kernels are allowed)
            hipMemset(devData + (nPow2 - 1), 0, sizeof(int));
            checkCUDAError("hipMemset root");

            // Down-sweep
            for (int d = levels - 1; d >= 0; --d) {
                int numOps = nPow2 >> (d + 1);
                dim3 block(BLOCK_SIZE);
                dim3 grid((numOps + BLOCK_SIZE - 1) / BLOCK_SIZE);
                kernDownSweep <<<grid, block >>> (nPow2, d, devData);
                checkCUDAError("kernDownSweep");
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            if (n <= 0) return;

            int nPow2 = 1 << ilog2ceil(n);

            int* devData = nullptr;
            hipMalloc(&devData, nPow2 * sizeof(int));
            checkCUDAError("hipMalloc");
            hipMemset(devData, 0, nPow2 * sizeof(int));
            checkCUDAError("hipMemset");

            hipMemcpy(devData, idata, n * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();
            scanInPlace(devData, nPow2); // exclusive scan in place
            timer().endGpuTimer();

            hipMemcpy(odata, devData, n * sizeof(int), hipMemcpyDeviceToHost);
            hipFree(devData);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            if (n <= 0) return 0;

            const int BLOCK_SIZE = 128;
            dim3 block(BLOCK_SIZE);
            dim3 grid((n + BLOCK_SIZE - 1) / BLOCK_SIZE);

            int* devIdata = nullptr, * devBools = nullptr, * devIndices = nullptr, * devOdata = nullptr;

            hipMalloc(&devIdata, n * sizeof(int));
            hipMalloc(&devBools, n * sizeof(int));
            hipMalloc(&devOdata, n * sizeof(int));
            checkCUDAError("hipMalloc inputs");
            hipMemcpy(devIdata, idata, n * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();

            // Map
            StreamCompaction::Common::kernMapToBoolean <<<grid, block >>> (n, devBools, devIdata);
            checkCUDAError("kernMapToBoolean");

            // Exclusive scan
            int nPow2 = 1 << ilog2ceil(n);
            hipMalloc(&devIndices, nPow2 * sizeof(int));
            checkCUDAError("hipMalloc devIndices");
            hipMemset(devIndices, 0, nPow2 * sizeof(int));
            checkCUDAError("hipMemset devIndices");
            hipMemcpy(devIndices, devBools, n * sizeof(int), hipMemcpyDeviceToDevice);
            checkCUDAError("D2D bools->indices");

            scanInPlace(devIndices, nPow2);

            // Scatter
            StreamCompaction::Common::kernScatter <<<grid, block >>> (n, devOdata, devIdata, devBools, devIndices);
            checkCUDAError("kernScatter");

            timer().endGpuTimer();

            int lastIdx = 0, lastFlag = 0;
            hipMemcpy(&lastIdx, devIndices + (n - 1), sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&lastFlag, devBools + (n - 1), sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("D2H count");
            int count = lastIdx + lastFlag;

            if (count > 0) {
                hipMemcpy(odata, devOdata, count * sizeof(int), hipMemcpyDeviceToHost);
                checkCUDAError("D2H odata");
            }

            hipFree(devIdata);
            hipFree(devBools);
            hipFree(devIndices);
            hipFree(devOdata);

            return count;
        }
    }
}
